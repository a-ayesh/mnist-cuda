
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void checkCudaKernel() {
    printf("CUDA kernel is running on the GPU!\n");
}

int main() {
    // Check CUDA Device Properties
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        cout << "No CUDA-compatible device detected.\n";
        return 1;
    } else {
        cout << "CUDA-compatible device found.\n";
    }

    // Launch a kernel to verify execution on the GPU
    checkCudaKernel<<<1, 1>>>();
    hipDeviceSynchronize(); // Ensure kernel execution completes

    return 0;
}
